#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2024, Shenzhen University
 * Immersive Media Laboratory, Institute of Future Media Computing
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use.
 */

#include <hip/hip_cooperative_groups.h>
#include "octree_builder.h"

#define BRANCH 2
#define THREADS 256

__device__ bool isEq3D(float4 p, float4 q)
{
    return (p.x == q.x && p.y == q.y && p.z == q.z);
}

__device__ bool isEq3D(float3 p, float3 q)
{
    return (p.x == q.x && p.y == q.y && p.z == q.z);
}

__device__ bool isEq3D(int3 p, int3 q)
{
    return (p.x == q.x && p.y == q.y && p.z == q.z);
}

__global__ void catchChildren(const int3* grid_coords, const int* levels, const int P, int* rst, const int lv)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= P || idx < 0) return;
    if (levels[idx] != lv) return;
    
    const int parent_level = lv - 1;

    int3 anchor = grid_coords[idx];

    int3 parent = {anchor.x / BRANCH, anchor.y / BRANCH, anchor.z / BRANCH};

    for (int i = 0; i < P; i++)
    {
        if (isEq3D(grid_coords[i], parent) && levels[i] == parent_level)
        {
            rst[idx] = i;
            return;
        }
    }
}

torch::Tensor buildOctree(const torch::Tensor& grid_coords, const torch::Tensor& levels)
{

    // catch cuda error
    hipError_t cudaStatus;

    const int P = grid_coords.size(0);
    const int levelMin = torch::min(levels).item().toInt();
    const int levelMax = torch::max(levels).item().toInt();

    // initialize `rst` for result
    auto rst_opts = grid_coords.options().dtype(torch::kInt32);
    torch::Tensor rst = torch::full({P}, -1, rst_opts);

    if (rst.device() != grid_coords.device() || grid_coords.device() != levels.device())
    {
        throw "Tensors should be on a same device.\n";
    }
    else
    {
        std::cout << "Starting Octree Matching" << std::endl;
    }

    // parallel computing
    int3* grid_coord_ptr;
    hipMalloc(&grid_coord_ptr, grid_coords.numel() * sizeof(int));
    hipMemcpy(grid_coord_ptr, grid_coords.contiguous().data_ptr<int>(), grid_coords.numel() * sizeof(int), hipMemcpyHostToDevice);

    int* level_ptr;
    hipMalloc(&level_ptr, levels.numel() * sizeof(int));
    hipMemcpy(level_ptr, levels.contiguous().data_ptr<int>(), levels.numel() * sizeof(int), hipMemcpyHostToDevice);
    int* rst_ptr = rst.contiguous().data_ptr<int>();
    hipDeviceSynchronize();

    int BLOCKS = (P + THREADS - 1) / THREADS;
    for (int lv = levelMax; lv > levelMin; lv--)
    {
        catchChildren<<<BLOCKS, THREADS>>>(grid_coord_ptr, level_ptr, P, rst_ptr, lv);
        hipDeviceSynchronize();
    }

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(cudaStatus));
    }
    hipDeviceSynchronize();

    return rst;
}